#include "hip/hip_runtime.h"
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

///////////////////////////////////////////////////////////////////////////////Operations
/*@
requires 0 <= n;
requires n < |xs|;
pure int get(seq<int> xs, int n) = xs[n];
@*/

/*@
requires 0 <= p;
ensures n == 2 ==> p < \result;
pure int exp(int n, int p) = 0 < p ? n * exp(n, p - 1) : 1;
@*/

/*@
requires n <= |xs|;
ensures n < 0 ==> |Take(xs, n)| == 0;
ensures 0 <= n ==> |Take(xs, n)| == n;
ensures (\forall int i; 0 <= i && i < n; xs[i] == get(Take(xs, n), i));
pure seq<int> Take(seq<int> xs, int n) =
	0 < n ? seq<int> { head(xs) } + Take(tail(xs), n - 1) : seq<int> { };
@*/

/*@
ensures |xs| == 0 ==> \result == 0;
ensures |xs| == 1 ==> \result == head(xs);
pure int intsum(seq<int> xs) =
	0 < |xs| ? head(xs) + intsum(tail(xs)) : 0;
@*/

/*@
requires 0 <= i && i <= |xs|;
ensures |\result| == |xs| - i;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, i+j+1))); 
pure seq<int> psum(seq<int> xs, int i) =
	i < |xs| ? seq<int> { intsum(Take(xs, i + 1)) } + psum(xs, i + 1) : seq<int> { };
@*/

// TODO use this version instead of the above `psum` (the above version is just a helper definition).
/*@
ensures |\result| == |xs|;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, j+1))); 
pure seq<int> psum2(seq<int> xs) = psum(xs, 0);
@*/

/*@
requires |input| >= 0;
requires i >= 0;
requires i <= |input|;
requires offset > 0;
requires offset <= |input|*2;
ensures |\result| == |input| - i;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(input, i + j + 1)) - intsum(Take(input, i + j + 1 - offset)));
pure seq<int> partial_prefixsum(seq<int> input, int i, int offset) = 	
i < |input| ? seq<int> {intsum(Take(input, i + 1)) - intsum(Take(input, i + 1 - offset))} + partial_prefixsum(input, i + 1, offset)	: seq<int> { };	
@*/

/////////////////////////////////////////////////////////////////////////////////////////////////Lemmas
/*@
requires |xs| <= 1;
requires i >= 0;
requires offset > 0;
requires offset <= |xs|;
ensures \result && (i < |xs| ==> xs == partial_prefixsum(xs, i, offset*2));
pure bool lemma_partial_prefixsum_base(seq<int> xs, int i, int offset) = true;
@*/	

/*@
requires 0 < n;
ensures \result && exp(2, n) == 2 * exp(2, n - 1);
pure bool lemma_exp2_red_mult(int n) = true;
@*/	

/*@
requires 0 < n;
ensures \result && exp(2, n) / 2 == exp(2, n - 1);
pure bool lemma_exp2_red_div(int n) = true;
@*/

/*@
requires 0 <= n;
ensures \result && 0 < exp(2, n);
pure bool lemma_exp2_positive(int n) =
  0 < n ? lemma_exp2_positive(n - 1) : true;
@*/

/*@
requires 0 <= i;
requires i <= j;
ensures \result && exp(2, i) <= exp(2, j);
pure bool lemma_exp2_leq(int i, int j) =
	0 < i ? lemma_exp2_leq(i - 1, j - 1) : lemma_exp2_positive(j);
@*/

/*@
requires 0 <= i && i < |xs|;
ensures \result && get(psum2(xs), i) == intsum(Take(xs, i+1));
pure bool lemma_psum_get(seq<int> xs, int i) =
  0 < |xs| ? (0 < i ? lemma_psum_get(tail(xs), i - 1) : true)
           : true;
@*/

/*@
requires j >= 0 && j <= |xs|;
ensures \result && (\forall int i; j <= i && i < |xs|; get(psum2(xs), i) == intsum(Take(xs, i+1)));
pure bool lemma_psum_get_all(seq<int> xs, int j) =
  j < |xs| ? lemma_psum_get(xs, j) && lemma_psum_get_all(xs, j+1) : true;
@*/

/*@
requires |xs| >= 0;
requires |ys| >= 0;
ensures \result && |xs| == 0 ==> intsum(xs + ys) == intsum(ys);
ensures \result && |ys| == 0 ==> intsum(xs + ys) == intsum(xs);
ensures \result && |xs + ys| == |xs| + |ys|;
ensures \result && intsum(tail(xs) + ys) == intsum(tail(xs)) + intsum(ys);
ensures \result && intsum(xs + ys) == intsum(xs) + intsum(ys);
pure bool lemma_intsum_app(seq<int> xs, seq<int> ys) =
  0 < |xs| ? lemma_intsum_app(tail(xs), ys) && tail(xs) + ys == tail(xs + ys) : true;
@*/

/*@
ensures \result && intsum(seq<int> { }) == 0;
pure bool lemma_intsum_zero() = true;
@*/

/*@
ensures \result && intsum(seq<int> { x }) == x;
pure bool lemma_intsum_single(int x) =
  tail(seq<int> { x }) == seq<int> { } && lemma_intsum_zero();
@*/

/*@
requires 0 <= n && n < |xs|;
ensures \result && Take(xs, n + 1) == Take(xs, n) + seq<int> { xs[n] };
pure bool missing_lemma_2(seq<int> xs, int n) =
  1 <= n ? missing_lemma_2(tail(xs), n - 1) : true;
@*/

/*@
requires |xs| >= 0;
requires i >= 0;
requires i < |xs|;
//ensures \result && (xs[i] == intsum(Take(xs, i + 1)) - intsum(Take(xs, i)));
pure bool lemma_intsum_Take(seq<int> xs, int i) =
	missing_lemma_2(xs, i) &&
	Take(xs, i + 1) == Take(xs, i) + seq<int> {xs[i]} &&
	lemma_intsum_app(Take(xs, i), seq<int> {xs[i]}) &&
	intsum( Take(xs, i) + seq<int> {xs[i]} ) == intsum(Take(xs, i)) + intsum(seq<int> {xs[i]}) &&
	intsum(Take(xs, i + 1)) == intsum(Take(xs, i)) + intsum(seq<int> {xs[i]}) &&
	lemma_intsum_single(xs[i]) &&
	xs[i] == intsum(seq<int> {xs[i]}) &&
  intsum(Take(xs, i + 1)) - intsum(Take(xs, i)) == intsum(Take(xs, i)) + intsum(seq<int> {xs[i]}) - intsum(Take(xs, i)) &&
  intsum(Take(xs, i + 1)) - intsum(Take(xs, i)) == intsum(seq<int> {xs[i]}) &&
  xs[i] == intsum(Take(xs, i + 1)) - intsum(Take(xs, i)) &&
  true;
@*/

////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@ 
//given seq<int> input_seq;
context_everywhere output != NULL;
context_everywhere k == 10;
//context_everywhere |input_seq| == exp(2, k);
context_everywhere opencl_gsize == exp(2, k);
context_everywhere opencl_gcount == 1;
//requires \pointer_index(output, \ltid, 1\2); 
//requires \ltid >= 1 ==> \pointer_index(output, \ltid - 1, 1\2); 
ensures \pointer_index(output, \ltid, 1\2);
@*/
__global__ void CUDA_Kernel_Kogge_Stone(int* output, int k)
{
  int tid = threadIdx.x;
  //@ assert tid == \ltid;
  
  //@ inhale \pointer_index(output, tid, 1\2); 
  //@ inhale tid >= 1 ==> \pointer_index(output, tid - 1, 1\2); 
  
  //@ ghost seq<int> out; 
  //@ assume |out| == exp(2, k);
  //@ assume output[tid] == out[tid];
  
  int offset = 1;
	int temp;
  //@ ghost seq<int> temp_seq = out; 
  //lemma_intsum_Take(temp_seq, tid);
	//@ assert missing_lemma_2(temp_seq, tid);
	//@ assert Take(temp_seq, tid + 1) == Take(temp_seq, tid) + seq<int> {temp_seq[tid]};
	//@ assert lemma_intsum_app(Take(temp_seq, tid), seq<int> {temp_seq[tid]});
	//@ assert intsum( Take(temp_seq, tid) + seq<int> {temp_seq[tid]} ) == intsum(Take(temp_seq, tid)) + intsum(seq<int> {temp_seq[tid]});
	//@ assert intsum(Take(temp_seq, tid + 1)) == intsum(Take(temp_seq, tid)) + intsum(seq<int> {temp_seq[tid]});
	//@ assert lemma_intsum_single(temp_seq[tid]);
	//@ assert temp_seq[tid] == intsum(seq<int> {temp_seq[tid]});
  
  //@ assert tid < offset ==> output[tid] == temp_seq[tid];
  
  //@ assert temp_seq[tid] == intsum(Take(temp_seq, tid + 1)) - intsum(Take(temp_seq, tid));

 
    
  /*@
	loop_invariant offset >= 1;
	loop_invariant |temp_seq| == |out|;
	loop_invariant offset < 2 * exp(2, k);
	loop_invariant \pointer_index(output, tid, 1\2); 
	loop_invariant tid >= offset ==> \pointer_index(output, tid - offset, 1\2); 
	loop_invariant temp_seq[tid] == intsum(Take(out, tid + 1)) - intsum(Take(out, tid + 1 - offset));
	loop_invariant tid < offset ==> temp_seq[tid] == intsum(Take(out, tid + 1));
	loop_invariant tid < offset ==> temp_seq[tid] == get(psum2(out), tid);
	loop_invariant temp_seq[tid] == get(partial_prefixsum(out, 0, offset), tid);
	loop_invariant tid < offset ==> output[tid] == temp_seq[tid]; 
  @*/
	while(offset < exp(2, k))
	{
		
		temp = output[tid];
		
		if (tid >= offset)
		{ 
			temp = temp + output[tid - offset];	
		}
		//@ assert tid < offset ==> temp == output[tid];
		//@ assert tid >= offset ==> temp == output[tid] + output[tid - offset];
		
     
    
    /*@
      context_everywhere |temp_seq| == exp(2, k);
			context tid >= 0 && tid < exp(2, k);
			context offset >= 1 && offset < exp(2, k);
			requires \pointer_index(output, tid, 1\2);
			requires tid >= offset ==> \pointer_index(output, tid - offset, 1\2);
			requires tid >= offset ==> temp == output[tid] + output[tid - offset];
      requires tid < offset  ==> output[tid] == temp_seq[tid];				
			ensures \pointer_index(output, tid, 1);	
      ensures tid < offset  ==> output[tid] == temp_seq[tid];	
    @*/
    __syncthreads();
    
   
		
		if (tid >= offset)
		{ 
			output[tid] = temp;
		}

    
		//@ assert tid < offset  ==> output[tid] == temp_seq[tid];
		//@ ghost temp_seq = partial_prefixsum(out, 0, offset * 2);
		//@ assert temp_seq[tid] == intsum(Take(out, tid + 1)) - intsum(Take(out, tid + 1 - offset * 2));
		//@ assert tid < offset * 2 ==> temp_seq[tid] == intsum(Take(out, tid + 1));
		//@ assert tid < offset * 2 ==> temp_seq[tid] == get(psum2(out), tid);
		//@ assert temp_seq[tid] == get(partial_prefixsum(out, 0, offset * 2), tid);
		
		
    
    
    /*@
      context_everywhere |temp_seq| == exp(2, k);
			context tid >= 0 && tid < exp(2, k);
			context offset >= 1 && offset < exp(2, k);
			requires \pointer_index(output, \ltid, 1);
      requires tid < offset ==> output[\ltid] == temp_seq[tid];
			ensures \pointer_index(output, tid, 1\2);
			ensures tid >= offset * 2 ==> \pointer_index(output, tid - offset * 2, 1\2);
      ensures tid < offset * 2 ==> output[tid] == temp_seq[tid]; 
    @*/
    __syncthreads(); 
    
		offset = offset * 2;
		
		
  }
	
	//@ assert temp_seq[tid] == get(psum2(out), tid);
	//@ assert output[tid] == temp_seq[tid];
	//@ assert output[tid] == get(psum2(out), tid);
	// assert input[tid] == out[tid];
  
  
  
}

////////////////////////////////////////////////////////////////////////////////
// CUDA Functions
////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, write);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, write);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
//@ context \pointer(src, N, read) ** \pointer(tgt, N, write);
//@ ensures (\forall int i; i >= 0 && i < N; src[i] == tgt[i]);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int CUDA_Host_Kogge_Stone( int argc, char** argv)
{
  int k = 10; // size of the input is 2^k
  
  int* host_input = vercorsMallocInt(exp(2, k)); // size of the host_input is 2^k
  int* host_output = vercorsMallocInt(exp(2, k)); // size of the host_output is 2^k
  
  //@ loop_invariant k == 10;
  //@ loop_invariant q >= 0 && q <= exp(2, k);
  //@ loop_invariant \pointer(host_input, exp(2, k), write) ** \pointer(host_output, exp(2, k), write);
  //@ loop_invariant (\forall int i; i >= 0 && i < q; host_input[i] == host_output[i]);
  for(int q=0; q<exp(2, k); q++)
  {
    host_output[q] = host_input[q];
  }
  
  //Copy the arrays to device memory
  int* device_output;
  device_output = vercorsCudaMallocInt(exp(2, k));
  //@ assert \pointer(device_output, exp(2, k), write);
  vercorsCudaMemcpyInt(device_output, host_output, exp(2, k), hipMemcpyHostToDevice) ;
  //@ assert (\forall int i; i >= 0 && i < exp(2, k); host_output[i] == device_output[i]);
  //@ assert (\forall int i; i >= 0 && i < exp(2, k); host_output[i] == host_input[i]);
  //@ assert (\forall int i; i >= 0 && i < exp(2, k); device_output[i] == host_input[i]);
  
  //setup execution parameters
	int num_of_blocks = 1;
	int num_of_threads_per_block = exp(2, k);

  
  //Kernel launch
  CUDA_Kernel_Kogge_Stone<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_output, k);
  //@ assert \pointer(device_output, exp(2, k), 1\2);
  //@ assert \pointer(host_input, exp(2, k), write);
  // copy result from device to host
  //vercorsCudaMemcpyInt(host_output, device_output, exp(2, k), hipMemcpyDeviceToHost);
  
  // cleanup memory
  vercorsFreeInt(host_output);
  vercorsCudaFreeInt(device_output);
  
}
