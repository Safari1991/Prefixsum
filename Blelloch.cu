#include "hip/hip_runtime.h"
/***********************************************************************************
Created by Mohsen Safari.
************************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>


////////////////////////////////////////////////////////////////////////////////
// Pure Functions
////////////////////////////////////////////////////////////////////////////////

/*@
requires 0 <= p;
ensures p < \result;
pure int ExpTwo(int p) = 0 < p ? 2 * ExpTwo(p - 1) : 1;
@*/

/*@
ensures |xs| == 0 ==> \result == 0;
ensures |xs| == 1 ==> \result == head(xs);
pure int intsum(seq<int> xs) =
	0 < |xs| ? head(xs) + intsum(tail(xs)) : 0;
@*/

/*@	
requires n <= |xs|;
ensures n < 0 ==> |Take(xs, n)| == 0;
ensures 0 <= n ==> |Take(xs, n)| == n;
ensures (\forall int i; 0 <= i && i < n; xs[i] == get(Take(xs, n), i));
pure seq<int> Take(seq<int> xs, int n) =
	0 < n ? seq<int> { head(xs) } + Take(tail(xs), n - 1) : seq<int> { };
@*/

/*@
requires 0 <= i && i <= |xs|;
ensures |\result| == |xs| - i;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, i+j))); 
pure seq<int> psum(seq<int> xs, int i) =
	i < |xs| ? seq<int> { intsum(Take(xs, i)) } + psum(xs, i + 1) : seq<int> { };
@*/

// TODO use this version instead of the above `psum` (the above version is just a helper definition).
/*@
ensures |\result| == |xs|;
ensures (\forall int j; 0 <= j && j < |\result|; \result[j] == intsum(Take(xs, j))); 
pure seq<int> psum2(seq<int> xs) = psum(xs, 0);
@*/

/*@	
requires |xs| >= 0;
ensures |xs| == 0	==> \result == xs;
ensures |xs| == 1 ==> \result == xs;
ensures |xs| == 2 ==> \result == seq<int> { head(xs) + head(tail(xs)) };
ensures |xs| % 2 == 0 ==> |\result| == |xs| / 2;
pure seq<int> implode(seq<int> xs) =
	1 < |xs| ? seq<int> { head(xs) + head(tail(xs)) } + implode(tail(tail(xs))) : xs;
@*/

/*@		
requires 0 <= p;
pure int exp(int n, int p) = 0 < p ? n * exp(n, p - 1) : 1;
@*/

/*@
requires 0 <= n;
requires n < |xs|;
pure int get(seq<int> xs, int n) = xs[n];		
@*/		

/*@			
requires k > 0;
requires |xs| == ExpTwo(k);
requires i >= 0 && i <= |xs|;
requires 1 <= lvl && lvl <= k;
requires stride == ExpTwo(lvl-1);	
requires stride > 0 && stride < |xs|;
ensures |\result| == |xs| - i;
ensures (\forall int j; j >= 0 && j < |\result|; ((i < |xs|) && ((i+j) >= stride) && (((i+j) % (2*stride)) == (2*stride-1))) ==> \result[j] == xs[i+j] + xs[i+j - stride]); 
ensures (\forall int j; j >= 0 && j < |\result|; ((i < |xs|) && (((i+j) < stride) || (((i+j) % (2*stride)) != (2*stride-1)))) ==> \result[j] == xs[i+j]);
pure seq<int> up(seq<int> xs, int stride, int i, int k, int lvl) =
	i < |xs| ? (
				((i % (2*stride)) == (2*stride-1) && (i >= stride)?
					seq<int> {xs[i] + xs[i-stride]} + up(xs, stride, i+1, k, lvl)
				:
					seq<int> {xs[i]} + up(xs, stride, i+1, k, lvl) ))
	:
		seq<int> {};
	
@*/	

////////////////////////////////////////////////////////////////////////////////////////Lemmas

/*@	
ensures \result && intsum(seq<int> { }) == 0;
pure bool lemma_intsum_zero() = true;
@*/

/*@
ensures \result && psum2(seq<int> { }) == seq<int> { };
pure bool lemma_psum_zero() = true;
@*/

/*@	
ensures \result && intsum(seq<int> { x }) == x;
pure bool lemma_intsum_single(int x);
@*/

/*@
requires |xs| == 1;
ensures \result && psum2(xs) == seq<int> {0};
pure bool lemma_psum_single(seq<int>  xs);
@*/

/*@
requires |xs| >= 0;
requires |ys| >= 0;
ensures |xs| == 0 ==> intsum(xs + ys) == intsum(ys);
ensures |ys| == 0 ==> intsum(xs + ys) == intsum(xs);
ensures |xs + ys| == |xs| + |ys|;
ensures \result && intsum(tail(xs) + ys) == intsum(tail(xs)) + intsum(ys);
ensures \result && intsum(xs + ys) == intsum(xs) + intsum(ys);
pure bool lemma_intsum_app(seq<int> xs, seq<int> ys);
@*/

/*@
requires |xs| <= 1;
ensures \result && xs == implode(xs);
pure bool lemma_implode_base(seq<int> xs) = true;
@*/


/*@
ensures \result && intsum(xs) == intsum(implode(xs));
pure bool lemma_implode_sum(seq<int> xs);
@*/

/*@
requires 0 < n;
ensures \result && ExpTwo(n) == 2 * ExpTwo(n - 1);
pure bool lemma_exp2_red_mult(int n)  = true;
@*/

/*@
requires 0 < n;
ensures \result && ExpTwo(n) / 2 == ExpTwo(n - 1);
pure bool lemma_exp2_red_div(int n) = true;
@*/

/*@
requires 0 <= n;
ensures \result && 0 < ExpTwo(n);
pure bool lemma_exp2_positive(int n);
@*/

/*@
requires 0 <= i;
requires i <= j;
ensures \result && ExpTwo(i) <= ExpTwo(j);
pure bool lemma_exp2_leq(int i, int j);
@*/

/*@
requires i >= 0 && j >= 0;
requires ExpTwo(i) == ExpTwo(j);
ensures \result && i == j;
pure bool power_two_lemma(int i, int j);
@*/

/*@
requires |xs| % 2 == 0;
ensures \result && |implode(xs)| == |xs| / 2;
pure bool lemma_implode_length_mod_two(seq<int> xs);
@*/

/*@
requires 0 < n && |xs| == ExpTwo(n);
ensures \result && |implode(xs)| == ExpTwo(n - 1);
pure bool lemma_implode_red_exp2(seq<int> xs, int n);
@*/

/*@
requires 0 < i;
requires i < |xs|;
ensures \result && get(tail(xs), i - 1) == xs[i];
pure bool lemma_intseq_index_tail(seq<int> xs, int i) = true;
@*/

/*@
requires |xs| % 2 == 0;
requires 0 <= i && i < |implode(xs)|;
requires (2 * i) < |xs|;
requires (2 * i + 1) < |xs|;
ensures \result && get(implode(xs), i) == xs[2 * i] + xs[2 * i + 1];
pure bool lemma_implode_get(seq<int> xs, int i);
@*/

/*@
requires j >= 0 && j <= |implode(xs)|;
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
ensures \result && (\forall int i; j <= i && i < |implode(xs)|; get(implode(xs), i) == xs[2 * i] + xs[2 * i + 1]);
pure bool lemma_implode_get_all(seq<int> xs, int j);
@*/  

/*@
requires |xs| == 2 * |ys|;
requires 0 <= |ys|;
requires (\forall int i; 0 <= i && i < |ys|; ys[i] == xs[2*i] + xs[2*i+1]);
ensures \result && ys == implode(xs);
pure bool lemma_implode_rel(seq<int> xs, seq<int> ys);
@*/

/*@
requires 0 <= i && i < |xs|;
ensures \result && get(psum2(xs), i) == intsum(Take(xs, i));
pure bool lemma_psum_get(seq<int> xs, int i);
@*/

/*@
requires j >= 0 && j <= |xs|;
ensures \result && (\forall int i; j <= i && i < |xs|; get(psum2(xs), i) == intsum(Take(xs, i)));
pure bool lemma_psum_get_all(seq<int> xs, int j);
@*/

/*@
requires 0 < n && n <= |xs|;
ensures \result && Take(xs, n) == Take(xs, n - 1) + seq<int> { xs[n - 1] };
pure bool missing_lemma_2(seq<int> xs, int n);
@*/

/*@
requires |xs| % 2 == 0;
requires |ys| % 2 == 0;
ensures \result && implode(xs + ys) == implode(xs) + implode(ys);
pure bool missing_lemma_3(seq<int> xs, seq<int> ys);
@*/

/*@
ensures \result && xs + (ys + zs) == (xs + ys) + zs;
pure bool intseq_concat_assoc(seq<int> xs, seq<int> ys, seq<int> zs) = true;
@*/

/*@
requires |xs| % 2 == 0;
requires 0 <= n && n < |implode(xs)|;
requires |implode(xs)| == |xs| / 2;
ensures \result && Take(implode(xs), n) == implode(Take(xs, 2 * n));
pure bool missing_lemma(seq<int> xs, int n);
@*/

/*@
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
requires 0 <= i && i < |implode(xs)|;
requires 2 * i < |xs|;
ensures \result && get(psum2(implode(xs)), i) == intsum(Take(xs, 2 * i));
pure bool lemma_psum_Take2(seq<int> xs, int i);
@*/


/*@
requires |xs| % 2 == 0;
requires |implode(xs)| == |xs|/2;
requires 0 <= i && i < |implode(xs)|;
requires 2 * i < |xs|;
ensures  \result && get(psum2(implode(xs)), i) == get(psum2(xs), 2 * i);
pure bool lemma_get_psum_implode(seq<int> xs, int i);
@*/

/*@
requires 0 <= i;
requires 2 * i + 1 < |xs|;
ensures  \result && get(psum2(xs), 2 * i + 1) == get(psum2(xs), 2 * i) + get(xs, 2 * i);
pure bool lemma_combine_psum(seq<int> xs, int i);
@*/


////////////////////////////////////////////////////////////////////////////////
//Kernel
////////////////////////////////////////////////////////////////////////////////
/*@ 
//given seq<int> input_seq;
context_everywhere output != NULL;
context_everywhere k == 10;
//context_everywhere |input_seq| == ExpTwo(k);
context_everywhere opencl_gsize == ExpTwo(k);
context_everywhere opencl_gcount == 1;
//requires (2 * \ltid  < ExpTwo(k)) ==> \pointer_index(output, 2 * \ltid, 1);
//requires (2 * \ltid + 1 < ExpTwo(k)) ==> \pointer_index(output, 2 * \ltid + 1, 1);
//ensures \pointer_index(output, \ltid, 1);
@*/
__global__ void CUDA_Kernel_Blelloch(int* output, int k)
{
  int tid = threadIdx.x;
  //@ assert tid == \ltid;
  
  //@ assume (2 * tid  < ExpTwo(k)) ==> \pointer_index(output, 2 * tid, 1);
  //@ assume (2 * tid + 1 < ExpTwo(k)) ==> \pointer_index(output, 2 * tid + 1, 1);

  //@ assume (tid == 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % 1 != 0; \pointer_index(output, i, 1));
  
  //@ ghost seq<int> input_seq; 
  //@ assume |input_seq| == ExpTwo(k);
	//@ assume (2 * tid  < ExpTwo(k)) ==> output[2 * tid] == input_seq[2 * tid];
	//@ assume (2 * tid + 1 < ExpTwo(k)) ==> output[2 * tid + 1] == input_seq[2 * tid + 1];
  
	int indicator = 2 * tid + 1;
	int stride = 1;

	int lvl = 1;
  
	//@ ghost seq<seq<int> > Matrix_UP = seq<seq<int> > { input_seq }; 
	//@ assert (\forall int i; 0 < i && i < lvl; Matrix_UP[i] == up(Matrix_UP[i - 1], stride/ExpTwo(lvl-i), 0, k, i));
	//@ ghost seq<seq<int> > Matrix = seq<seq<int> > { input_seq };
  
  /*@
	loop_invariant k > 0;
	loop_invariant tid >= 0 && tid < ExpTwo(k);
	loop_invariant stride > 0;
	loop_invariant 1 <= lvl;
	loop_invariant stride == ExpTwo(lvl-1);	
	loop_invariant lvl <= k+1;
	loop_invariant indicator + 1 == ExpTwo(lvl)*(tid+1);
	loop_invariant indicator + 1 == 2*stride*(tid+1);
	loop_invariant indicator > 0;
	loop_invariant stride <= ExpTwo(k);
	loop_invariant indicator < ExpTwo(k) ==> \pointer_index(output, indicator, 1); 
	loop_invariant indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(output, indicator - stride, 1); 
	loop_invariant tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(output, i, 1));
	loop_invariant (tid==0 && (stride == ExpTwo(k))) ==> \pointer_index(output, ExpTwo(k) - 1, 1);
	loop_invariant |Matrix_UP| == lvl;
	loop_invariant (\forall int i; 0 <= i && i < lvl; |Matrix_UP[i]| == ExpTwo(k));
	loop_invariant lvl == 1 ==> Matrix_UP[lvl - 1] == input_seq;
	loop_invariant lvl > 1 && lvl < |Matrix_UP| ==> Matrix_UP[lvl] == up(Matrix_UP[lvl - 1], (stride/2) - 1, 0, k, lvl - 1);
	loop_invariant indicator < ExpTwo(k) ==> Matrix_UP[lvl - 1][indicator] == output[indicator];
	loop_invariant indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator - stride] == output[indicator - stride];
	loop_invariant lvl == k+1 ==> Matrix_UP[lvl-1][ExpTwo(k) - 1] == intsum(input_seq);
	loop_invariant lvl == k+1 ==> Matrix_UP[lvl-1][(ExpTwo(k) - 1)/2] == intsum(Take(input_seq, |input_seq|/2)); 
	loop_invariant |Matrix| == lvl;
	loop_invariant (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k)); 
	loop_invariant (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
	loop_invariant (\forall int i; 0 < i && i < lvl; Matrix[i] == implode(Matrix[i - 1]));
	loop_invariant (\forall int i; 0 <= i && i < lvl; intsum(Matrix[i]) == intsum(input_seq));
	loop_invariant Matrix[0] == input_seq;
	loop_invariant indicator < ExpTwo(k) && 2 * tid + 1 < |Matrix[lvl - 1]| ==> output[indicator] == Matrix[lvl - 1][2 * tid + 1];
	loop_invariant indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> output[indicator - stride] == Matrix[lvl - 1][2 * tid];
	@*/
  while(stride < ExpTwo(k))
	{
		
		
		if(indicator < ExpTwo(k) && indicator >= stride)
		{
			//@ assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> output[indicator] == Matrix[lvl - 1][2 * tid + 1];
			//@ assert 2 * tid < |Matrix[lvl - 1]| ==> output[indicator - stride] == Matrix[lvl - 1][2 * tid];
			output[indicator] = output[indicator] + output[indicator - stride];
			//@ assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> output[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid]; 
		}
		
		//@ assert lemma_implode_length_mod_two(Matrix[lvl - 1]);
		//@ assert lemma_implode_sum(Matrix[lvl - 1]);
		//@ assert lemma_implode_get_all(Matrix[lvl - 1], 0);
		

		//@ ghost Matrix = Matrix + seq<seq<int> > { implode(Matrix[lvl - 1]) };
		
		//@ ghost tid < |implode(Matrix[lvl - 1])| ? (lemma_implode_get(Matrix[lvl - 1], tid) && (2 * tid + 1 < |Matrix[lvl - 1]| ==> get(implode(Matrix[lvl - 1]), tid) == Matrix[lvl - 1][2 * tid] + Matrix[lvl - 1][2 * tid + 1]) && (indicator < ExpTwo(k) && indicator >= stride ==> output[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid]) && (Matrix[lvl] == implode(Matrix[lvl - 1])) && (indicator < ExpTwo(k) && indicator >= stride ==> output[indicator] == Matrix[lvl][tid])) : true;
    
    
                                               
    
    /*if(tid < |implode(Matrix[lvl - 1])|){
		lemma_implode_get(Matrix[lvl - 1], tid);
		assert 2 * tid + 1 < |Matrix[lvl - 1]| ==> get(implode(Matrix[lvl - 1]), tid) == Matrix[lvl - 1][2 * tid] + Matrix[lvl - 1][2 * tid + 1];
		assert indicator < output.length && indicator >= stride ==> output[indicator] == Matrix[lvl - 1][2 * tid + 1] + Matrix[lvl - 1][2 * tid];
		assert Matrix[lvl] == implode(Matrix[lvl - 1]);
		assert indicator < output.length && indicator >= stride ==> output[indicator] == Matrix[lvl][tid];
		}*/
		
    
    /*@
			context_everywhere k > 0;
			context_everywhere 1 <= lvl && lvl <= k;	
			context_everywhere |Matrix| == lvl + 1;
			requires tid >= 0 && tid < ExpTwo(k);
			requires stride == ExpTwo(lvl-1);
			requires stride > 0 && stride < ExpTwo(k);
			requires indicator + 1 == ExpTwo(lvl)*(tid+1);
			requires indicator + 1 == 2*stride*(tid+1);
			requires indicator > 0;
			requires indicator < ExpTwo(k) ==> \pointer_index(output, indicator, 1);
			requires indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(output, indicator - stride, 1);
			requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(output, i, 1));
			ensures tid >= 0 && tid < ExpTwo(k);
			ensures 2 * stride == ExpTwo(lvl);
			ensures 2 * stride > 0 && 2 * stride <= ExpTwo(k);
			ensures 2 * indicator + 2 == ExpTwo(lvl+1)*(tid+1);
			ensures 2 * indicator + 2 == 2*stride*(tid+1);
			ensures 2 * indicator + 1 > 0;
			ensures 2 * indicator + 1 < ExpTwo(lvl) ==> \pointer_index(output, 2 * indicator + 1, 1);
			ensures 2 * indicator + 1 < ExpTwo(lvl) && 2 * indicator + 1 >= 2 * stride  ==> \pointer_index(output, 2 * indicator + 1 - 2 * stride, 1);
			ensures tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(lvl) && (i + 1) % (2 * stride) != 0; \pointer_index(output, i, 1));
			ensures (tid==0 && (2 * stride == ExpTwo(lvl))) ==> \pointer_index(output, ExpTwo(k) - 1, 1);
    @*/
    __syncthreads();
		
		//@ ghost Matrix_UP = Matrix_UP + seq<seq<int> > { up(Matrix_UP[lvl - 1], stride, 0, k, lvl) };
		//@ assert (indicator < ExpTwo(k)) && (indicator >= stride) ==> Matrix_UP[lvl][indicator] == Matrix_UP[lvl - 1][indicator] + Matrix_UP[lvl - 1][indicator-stride]; 
		indicator = 2 * indicator + 1;
		stride = 2 * stride;	
		lvl = lvl + 1;
		//@ assert (\forall int i; 0 < i && i < lvl; Matrix_UP[i] == up(Matrix_UP[i - 1], stride/ExpTwo(lvl-i), 0, k, i));
		
		//@ assert stride == ExpTwo(lvl-1);
		//@ assert lemma_exp2_red_mult(lvl);
		//@ assert ExpTwo(lvl) == 2 * ExpTwo(lvl - 1);
		//@ assert 2*stride == ExpTwo(lvl);
		//@ assert indicator + 1 == ExpTwo(lvl)*(tid+1);
		//@ assert indicator + 1 == 2*stride*(tid+1);

		
	}
  
	//@ assert stride == ExpTwo(lvl-1);
	//@ assert ExpTwo(lvl-1) == ExpTwo(k); 
	//@ assert stride == ExpTwo(k);
	//@ assert power_two_lemma(lvl-1, k);
	//@ assert lvl == k + 1;
	//@ assert indicator < ExpTwo(k) ==> Matrix_UP[lvl - 1][indicator] == output[indicator];
	//@ assert |Matrix| == lvl;
	//@ assert (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
	//@ assert (\forall int i; 0 < i && i < k + 1; Matrix[i] == implode(Matrix[i - 1]));
	//@ assert (\forall int i; 0 <= i && i < k + 1; intsum(Matrix[i]) == intsum(input_seq));
	//@ assert |Matrix[k]| == 1;
	//@ assert lemma_intsum_single(Matrix[k][0]);
	//@ assert intsum(Matrix[k]) == intsum(input_seq);
	//@ assert Matrix[k] == seq<int>{intsum(input_seq)};
	//@ assert Matrix[0] == input_seq;
	//@ assert (\forall int i; 0 <= i && i < k + 1; 0 < |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
  
/////////////////////////////////////////////////////////////////////////////////	

  //@ assert indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> output[indicator - stride] == Matrix[lvl - 1][2 * tid];

  /*@
		context_everywhere k > 0;
		context_everywhere |Matrix_UP| == k + 1;
    context_everywhere |Matrix| == k + 1;
		context_everywhere lvl == k + 1;
		context stride == ExpTwo(k);
		context indicator + 1 == ExpTwo(lvl)*(tid+1);
		context indicator + 1 == 2*stride*(tid+1);
		context indicator > 0;
		context stride > 0 ;
		requires indicator < ExpTwo(k) ==> \pointer_index(output, indicator, 1);
		requires indicator < ExpTwo(k) && indicator >= stride  ==> \pointer_index(output, indicator - stride, 1);
		requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(output, i, 1));
		requires (tid==0 && (stride == ExpTwo(k))) ==> \pointer_index(output, ExpTwo(k) - 1, 1);
		requires (\forall int i; 0 <= i && i <= k; |Matrix_UP[i]| == ExpTwo(k));
    requires (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
    requires (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k)); 
		requires indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator] == output[indicator];
		requires indicator < ExpTwo(k) && indicator >= stride ==> Matrix_UP[lvl - 1][indicator - stride] == output[indicator - stride];
    requires indicator < ExpTwo(k) && indicator >= stride && 2 * tid < |Matrix[lvl - 1]| ==> output[indicator - stride] == Matrix[lvl - 1][2 * tid];
		context tid >= 0 && tid < ExpTwo(k);
		//ensures stride == ExpTwo(k) / 2;
		//ensures indicator == ExpTwo(k) * tid + ExpTwo(k) - 1;
		//ensures stride > 0 ;
		//ensures indicator > 0;
		ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) ==> \pointer_index(output, ExpTwo(k) * \ltid + ExpTwo(k) - 1, 1);
		ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2  ==> \pointer_index(output, ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2, 1);
		ensures tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % (ExpTwo(k) / 2) != 0; \pointer_index(output, i, 1));
    ensures (\forall int i; 0 <= i && i <= k; |Matrix_UP[i]| == ExpTwo(k));
    ensures (\forall int i; 0 <= i && i < lvl; |Matrix[i]| == ExpTwo(k - i));
    ensures (\forall int i; 0 <= i && i < lvl; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k)); 
		//ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl - 1][ExpTwo(k) * \ltid + ExpTwo(k) - 1] == output[ExpTwo(k) * \ltid + ExpTwo(k) - 1];
		//ensures ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl - 1][ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == output[ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2];		
    //ensures 2 * tid < |Matrix[lvl-2]| && ExpTwo(k) * \ltid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * \ltid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> output[ExpTwo(k) * \ltid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == get(Matrix[lvl-2], 2 * tid);  
  @*/
  __syncthreads();
    
  // (unstability) These come from the last three postconditions in the previous barrier:
		//@ assume ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl - 1][ExpTwo(k) * tid + ExpTwo(k) - 1] == output[ExpTwo(k) * tid + ExpTwo(k) - 1];
		//@ assume ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl - 1][ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == output[ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2];		
    //@ assume 2 * tid < |Matrix[lvl-2]| && ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> output[ExpTwo(k) * tid + ExpTwo(k) - 1 - ExpTwo(k) / 2] == get(Matrix[lvl-2], 2 * tid);  
  
///////////////////////////////////////////////////////////////////////////////////////	Down		

	indicator = ExpTwo(k) * tid + ExpTwo(k) - 1; // output.length * tid + output.length - 1;
	stride = ExpTwo(k) / 2; // output.length / 2;
	lvl = k - 1; //lvl - 2;
	int temp;
	//@ ghost seq<int> temp_seq = seq<int> { 0 };
	
	//@ assert ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) ==> Matrix_UP[lvl + 1][indicator] == output[indicator];
	//@ assert ExpTwo(k) * tid + ExpTwo(k) - 1 < ExpTwo(k) && ExpTwo(k) * tid + ExpTwo(k) - 1 >= ExpTwo(k) / 2 ==> Matrix_UP[lvl + 1][indicator - stride] == output[indicator - stride];

 
	
	if(indicator < ExpTwo(k))
	{
		output[indicator] = 0;
	}
  
	

  
  
  /*@
  loop_invariant k > 0;
  loop_invariant tid >= 0 && tid < ExpTwo(k);
  loop_invariant lvl <= k - 1;
  loop_invariant lvl >= -1;
  loop_invariant lvl >= 0 ==> stride == ExpTwo(lvl);
  loop_invariant lvl == -1 ==> stride == 0;
  loop_invariant stride == 0 ==> lvl == -1;
  loop_invariant stride >= 0;
  loop_invariant indicator >= 0;
  loop_invariant indicator+1 == ExpTwo(lvl+1)*(tid+1);
  loop_invariant indicator < ExpTwo(k) ==> \pointer_index(output, indicator, 1);
  loop_invariant lvl >= 0 && indicator < ExpTwo(k) && indicator >= stride ==> \pointer_index(output, indicator - stride, 1); 
  loop_invariant (tid==0 && stride > 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(output, i, 1));
  //loop_invariant lvl == -1 ==> \pointer_index(output, tid, 1);
  //loop_invariant lvl == -1 ==> indicator == tid;
  //loop_invariant indicator == tid ==> lvl == -1;
  loop_invariant |temp_seq| == ExpTwo(k - (lvl + 1));
  loop_invariant 0 < |temp_seq| && |temp_seq| <= ExpTwo(k);
  loop_invariant temp_seq == psum2(Matrix[lvl + 1]);
  loop_invariant (\forall int i; 0 <= i && i < k + 1; 0 < |Matrix[i]| && |Matrix[i]| <= ExpTwo(k));
  loop_invariant (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
  loop_invariant (\forall int i; 0 <= i && i < k + 1; intsum(Matrix[i]) == intsum(input_seq));
  loop_invariant (\forall int i; 0 < i && i < k + 1; Matrix[i] == implode(Matrix[i - 1])); 
  loop_invariant Matrix[0] == input_seq;
  loop_invariant Matrix[k] == seq<int>{ intsum(input_seq) };
  loop_invariant tid < |temp_seq| && indicator < ExpTwo(k) ==> temp_seq[tid] == output[indicator];
  loop_invariant lvl >= 0 && 2 * tid < |Matrix[lvl]| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator - stride] == get(Matrix[lvl], 2 * tid);
  @*/
  while(stride >= 1)
  {
  	if(indicator < ExpTwo(k) && indicator >= stride)
  	{
		
  		//@ assert tid < |temp_seq| ==> temp_seq[tid] == output[indicator];
  		temp = output[indicator];
  		//@ assert tid < |temp_seq| ==> temp == temp_seq[tid];
  		output[indicator] = output[indicator] + output[indicator - stride];
  		//@ assert tid < |temp_seq| ==> output[indicator] == temp_seq[tid] + output[indicator - stride];
	  
  		//@ assert 2 * tid < |Matrix[lvl]| ==> output[indicator - stride] == get(Matrix[lvl], 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| && tid < |temp_seq| ==> output[indicator] == temp_seq[tid] + get(Matrix[lvl], 2 * tid);
  		//@ assert tid < |Matrix[lvl + 1]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl + 1]), tid); 
  		//@ assert tid < |Matrix[lvl + 1]| && 2 * tid < |Matrix[lvl]| ==> output[indicator] == get(psum2(Matrix[lvl + 1]), tid) + get(Matrix[lvl], 2 * tid); 
  		//@ assert Matrix[lvl + 1] == implode(Matrix[lvl]);
  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> output[indicator] == get(psum2(implode(Matrix[lvl])), tid) + get(Matrix[lvl], 2 * tid);	
      //@ ghost tid < |implode(Matrix[lvl])| ? lemma_get_psum_implode(Matrix[lvl], tid) : true;
  		/*if(tid < |implode(Matrix[lvl])|){
  			lemma_get_psum_implode(Matrix[lvl], tid);
  		}*/												 
  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> get(psum2(implode(Matrix[lvl])), tid) == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| ==> output[indicator] == get(psum2(Matrix[lvl]), 2 * tid) + get(Matrix[lvl], 2 * tid);
  		//@ ghost 2 * tid + 1 < |Matrix[lvl]| ? lemma_combine_psum(Matrix[lvl], tid) : true;
      /*if(2 * tid + 1 < |Matrix[lvl]|){
  		  lemma_combine_psum(Matrix[lvl], tid);
  		}*/
  		//@ assert 2 * tid + 1 < |Matrix[lvl]| ==> get(psum2(Matrix[lvl]), 2 * tid + 1) == get(psum2(Matrix[lvl]), 2 * tid) + get(Matrix[lvl], 2 * tid);                      		
  		//@ assert 2 * tid + 1 < |Matrix[lvl]| ==> output[indicator] == get(psum2(Matrix[lvl]), 2 * tid + 1);
		
  		//@ assert tid < |temp_seq| ==> temp == temp_seq[tid];
  		output[indicator - stride] = temp;
  		//@ assert tid < |temp_seq| ==> output[indicator - stride] == temp_seq[tid];
		
  		//@ assert tid < |Matrix[lvl + 1]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl + 1]), tid);
  		//@ assert Matrix[lvl + 1] == implode(Matrix[lvl]); 
  		//@ assert tid < |implode(Matrix[lvl])| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(implode(Matrix[lvl])), tid);
  		//@ ghost tid < |implode(Matrix[lvl])| ? lemma_get_psum_implode(Matrix[lvl], tid) : true;
      /*if(tid < |implode(Matrix[lvl])|){
  			lemma_get_psum_implode(Matrix[lvl], tid);
  		}*/
    
  		//@ assert tid < |implode(Matrix[lvl])| && 2 * tid < |Matrix[lvl]| ==> get(psum2(implode(Matrix[lvl])), tid) == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| && tid < |temp_seq| ==> temp_seq[tid] == get(psum2(Matrix[lvl]), 2 * tid);
  		//@ assert 2 * tid < |Matrix[lvl]| ==> output[indicator - stride] == get(psum2(Matrix[lvl]), 2 * tid);
		
  	}
	
  	//@ ghost temp_seq = psum2(Matrix[lvl]);  

  	//@ assert 2 * tid < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator - stride] == temp_seq[2 * tid];
  	//@ assert 2 * tid + 1 < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator] == temp_seq[2 * tid + 1]; 
  
  
    /*@
  		context_everywhere lvl >= 0 && lvl <= k - 1;
  		requires tid >= 0 && tid < ExpTwo(k);
      context_everywhere |temp_seq| == ExpTwo(k - lvl);
      context_everywhere 0 < |temp_seq| && |temp_seq| <= ExpTwo(k);
      context_everywhere |Matrix| == k + 1;
      //context lvl - 1 == -1 ==> (indicator - 1) / 2 == \ltid;
      //context (indicator - 1) / 2 == \ltid ==> lvl - 1 == -1;
  		requires indicator >= 0;
  		requires stride >= 1 ;
  		requires stride == ExpTwo(lvl);
  		requires indicator+1 == ExpTwo(lvl+1)*(\ltid+1);
  		requires indicator < ExpTwo(k) ==> \pointer_index(output, indicator, 1);
  		requires indicator < ExpTwo(k) && indicator >= stride  ==> \pointer_index(output, indicator - stride, 1); 
  		requires tid==0 ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % stride != 0; \pointer_index(output, i, 1));
      //requires 2 * tid < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator - stride] == temp_seq[2 * tid];
      //requires 2 * tid + 1 < |temp_seq| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator] == temp_seq[2 * tid + 1]; 
      requires (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
      requires (\forall int i; 0 <= i && i < k + 1; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k)); 
      //requires 2 * tid < |Matrix[lvl]| && indicator < ExpTwo(k) && indicator >= stride ==> output[indicator - stride] == get(psum2(Matrix[lvl]), 2 * tid);
  		ensures tid >= 0 && tid < ExpTwo(k);
  		ensures lvl-1 >= 0 ==> stride / 2 == ExpTwo(lvl - 1);
  		ensures lvl-1 == -1 ==> stride / 2 == 0;
      ensures stride / 2 == 0  ==> lvl-1 == -1;
  		ensures stride / 2 >= 0;
  		ensures (indicator - 1) / 2 >= 0;
  		ensures (indicator - 1) / 2+1 == ExpTwo(lvl)*(tid+1);
  		ensures (indicator - 1) / 2 < ExpTwo(k) ==> \pointer_index(output, (indicator - 1) / 2, 1);
  		ensures lvl-1 >= 0 && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2  ==> \pointer_index(output, (indicator - 1) / 2 - stride / 2, 1);
  		ensures (tid==0 && stride/2 > 0) ==> (\forall* int i; 0 <= i && i < ExpTwo(k) && (i + 1) % (stride/2) != 0; \pointer_index(output, i, 1));
      ensures (\forall int i; 0 <= i && i < k + 1; |Matrix[i]| == ExpTwo(k - i));
      ensures (\forall int i; 0 <= i && i < k + 1; 0 <= |Matrix[i]| && |Matrix[i]| <= ExpTwo(k)); 
      //ensures tid < |temp_seq| && (indicator - 1) / 2 < ExpTwo(k) ==> temp_seq[tid] == output[(indicator - 1) / 2];
      //ensures lvl-1 >= 0 && 2 * tid < |Matrix[lvl-1]| && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2 ==> output[(indicator - 1) / 2 - stride / 2] == get(Matrix[lvl-1], 2 * tid);
    @*/
    __syncthreads();
    
    // (unstability) These come from the last two postconditions in the previous barrier:
  	//@ assume tid < |temp_seq| && (indicator - 1) / 2 < ExpTwo(k) ==> temp_seq[tid] == output[(indicator - 1) / 2];
  	//@ assume lvl-1 >= 0 && 2 * tid < |Matrix[lvl-1]| && (indicator - 1) / 2 < ExpTwo(k) && (indicator - 1) / 2 >= stride / 2 ==> output[(indicator - 1) / 2 - stride / 2] == get(Matrix[lvl-1], 2 * tid);
  	
  	indicator = (indicator - 1) / 2;
  	stride = stride / 2;
  	lvl = lvl - 1;  
    

  }


  //@ assert temp_seq == psum2(Matrix[0]);
  //@ assert Matrix[0] == input_seq;
  //@ assert temp_seq == psum2(input_seq);
  //@ assert tid < |temp_seq| && indicator < ExpTwo(k) ==> temp_seq[tid] == output[indicator];

  
  
 

}


////////////////////////////////////////////////////////////////////////////////
// CUDA Functions
////////////////////////////////////////////////////////////////////////////////
//@ ensures \pointer(\result, N, 1);
int *vercorsMallocInt(int N);
void vercorsFreeInt(int *ar);
//@ ensures \pointer(\result, N, 1);
int *vercorsCudaMallocInt(int N);
void vercorsCudaFreeInt(int *addr);
//@ context \pointer(src, N, read) ** \pointer(tgt, N, 1);
//@ ensures (\forall int i; i >= 0 && i < N; src[i] == tgt[i]);
void vercorsCudaMemcpyInt(int *tgt, int *src, int N, int direction); 


////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int CUDA_Host_Blelloch( int argc, char** argv)
{
  int k = 10; // size of the input is 2^k
  
  int* host_input = vercorsMallocInt(ExpTwo(k)); // size of the host_input is 2^k
  int* host_output = vercorsMallocInt(ExpTwo(k)); // size of the host_output is 2^k
  
  //@ loop_invariant k == 10;
  //@ loop_invariant q >= 0 && q <= ExpTwo(k);
  //@ loop_invariant \pointer(host_input, ExpTwo(k), 1) ** \pointer(host_output, ExpTwo(k), 1);
  //@ loop_invariant (\forall int i; i >= 0 && i < q; host_input[i] == host_output[i]);
  for(int q=0; q<ExpTwo(k); q++)
  {
    host_output[q] = host_input[q];
  }
  
  //Copy the arrays to device memory
  int* device_output;
  device_output = vercorsCudaMallocInt(ExpTwo(k));
  vercorsCudaMemcpyInt(device_output, host_output, ExpTwo(k), hipMemcpyHostToDevice) ;
  //@ assert (\forall int i; i >= 0 && i < ExpTwo(k); host_output[i] == device_output[i]);
  //@ assert (\forall int i; i >= 0 && i < ExpTwo(k); host_output[i] == host_input[i]);
  //@ assert (\forall int i; i >= 0 && i < ExpTwo(k); device_output[i] == host_input[i]);
  
  //setup execution parameters
	int num_of_blocks = 1;
	int num_of_threads_per_block = ExpTwo(k);
  
  

  
  //Kernel launch
  CUDA_Kernel_Blelloch<<< /*grid*/num_of_blocks, /*threads*/num_of_threads_per_block/*, 0*/ >>>(device_output, k);
  
  // copy result from device to host
  //vercorsCudaMemcpyInt(host_output, device_output, ExpTwo(k), hipMemcpyDeviceToHost);
  
  // cleanup memory
  vercorsFreeInt(host_output);
  vercorsCudaFreeInt(device_output);
  
}
